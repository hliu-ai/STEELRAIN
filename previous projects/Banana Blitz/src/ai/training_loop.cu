#include "hip/hip_runtime.h"
#include "training_loop.hpp"
#include "replay_buffer.hpp"
#include "ai/policy_network.hpp"
#include "ai/target_network.hpp"
#include "ai/input_handler.hpp"
#include <torch/torch.h>
#include <iostream>
#include <string>
#include <fstream>
#include <numeric>

    // Constructor implementation
TrainingLoop::TrainingLoop(float gamma, float learning_rate, PolicyNetwork& policy_network)
    : gamma(gamma),
      learning_rate(learning_rate),
      policy_network(policy_network), // Initialize policy network reference
      optimizer(policy_network.parameters(), torch::optim::AdamOptions(learning_rate)) { // Create optimizer
}

// Function to run one step of training
void TrainingLoop::train_step(
    TargetNetwork& target_network,
    ReplayBuffer& replay_buffer,
    size_t batch_size) {
    if (replay_buffer.size() < batch_size) {
        std::cout << "Not enough experiences in the replay buffer yet to sample!" << std::endl;
        return;
    }

    // Sample a batch from the replay buffer
    std::vector<std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>> batch = replay_buffer.sample_batch(batch_size);
    // Extract batched tensors from the replay buffer and move them to CUDA
    torch::Tensor batched_states = replay_buffer.get_batched_states();
    torch::Tensor batched_actions = replay_buffer.get_batched_actions();
    torch::Tensor batched_rewards = replay_buffer.get_batched_rewards();
    torch::Tensor batched_next_states = replay_buffer.get_batched_next_states();
    torch::Tensor batched_dones = replay_buffer.get_batched_dones();


    // Reset noise in the policy network before the forward pass
    policy_network.reset_noise_all();
    // Forward pass through the policy network for current Q-values
    this->batched_q_values = policy_network.forward(batched_states);
    this->batched_q_values_taken = batched_q_values.gather(1, batched_actions.to(torch::kLong));

    // Reset noise in the target network before computing the target Q-values
    target_network.reset_noise_all();
    // Forward pass through the target network to get next Q-values (detach to remove from computation graph)
    this->batched_next_q_targets = target_network.forward(batched_next_states).detach();
    
    // Forward pass through the policy network for next state Q-values (used for choosing the best action)
    //policy_network.reset_noise_all();  // Optionally refresh noise again if desired - i wont here.
    this->batched_next_q_policy = policy_network.forward(batched_next_states);
    this->batched_next_actions = std::get<1>(batched_next_q_policy.max(1, true));
    this->batched_selected_q_next_values = batched_next_q_targets.gather(1, batched_next_actions);

    // Bellman Equation: compute target Q-values
    this->target_q_values = batched_rewards + (gamma * batched_selected_q_next_values * (1 - batched_dones));

    // Compute the loss using MSE between current and target Q-values
    this->loss = torch::mse_loss(batched_q_values_taken, target_q_values);

    optimizer.zero_grad();  // Clear old gradients
    loss.backward();        // Compute gradients via backpropagation
    optimizer.step();       // Update the policy network's weights

    // Update the loss buffer for logging
    loss_buffer.push_back(loss.item<float>());
    if (loss_buffer.size() > max_loss_buffer_size) {
        loss_buffer.pop_front();
    }
}

    /*
    // Print debug information
    std::cout << "Training step completed. Loss: " << loss.item<float>() << std::endl;
    std::cout << "Loss Buffer: ";
    for (const auto& loss : loss_buffer) {
        std::cout << loss << " ";
    }
    std::cout << std::endl;
    std::cout << "Average Loss: " << get_average_loss() << std::endl;
    */

void TrainingLoop::save_loss_to_csv(const std::string& file_path, size_t step) const {
    std::ofstream file(file_path, std::ios::app);  // Open file in append mode
    if (file.is_open()) {
        // Calculate average loss from the rolling buffer
        float average_loss = std::accumulate(loss_buffer.begin(), loss_buffer.end(), 0.0f) / loss_buffer.size();

        // Write the current step and averaged loss to CSV
        file << step << "," << average_loss << "\n";
    } else {
        std::cerr << "Error: Could not open file for writing loss data.\n";
    }
}


// Getter implementations
torch::Tensor TrainingLoop::get_batched_next_q_targets() const {
    return batched_next_q_targets;
}

torch::Tensor TrainingLoop::get_batched_next_q_policy() const {
    return batched_next_q_policy;
}

torch::Tensor TrainingLoop::get_batched_next_actions() const {
    return batched_next_actions;
}

torch::Tensor TrainingLoop::get_batched_selected_q_next_values() const {
    return batched_selected_q_next_values;
}

torch::Tensor TrainingLoop::get_batched_q_values() const {
    return batched_q_values;
}

torch::Tensor TrainingLoop::get_batched_q_values_taken() const {
    return batched_q_values_taken;
}

torch::Tensor TrainingLoop::get_target_q_values() const {
    return target_q_values;
}

torch::Tensor TrainingLoop::get_loss() const {
    return loss;
}

float TrainingLoop::get_average_loss() const {
    if (loss_buffer.empty()) {
        return 0.0f; //no loss values recorded yet
    }
    return std::accumulate(loss_buffer.begin(), loss_buffer.end(), 0.0f) / loss_buffer.size();
}

const std::deque<float>& TrainingLoop::get_recent_losses() const {
    return loss_buffer; //provide access to the raw loss values
}

void TrainingLoop::copy_weights_from_policy_to_target(PolicyNetwork& policy_network,
                                                      TargetNetwork& target_network,
                                                      InputHandler& input_handler,
                                                      int global_frame_counter,
                                                      int copy_interval) {
    // Check if it's time to copy weights
    if (global_frame_counter % copy_interval == 0) {
        // Copy weights
        target_network.copy_weights_from(policy_network);
        std::cout << "Target network updated at frame interval: " << global_frame_counter << "." << std::endl;

        // Generate debugging input tensor
        torch::Tensor debugging_input_tensor = input_handler.generate_debug_input();

        // Pass through both networks
        torch::Tensor policy_output = policy_network.forward(debugging_input_tensor);
        torch::Tensor target_output = target_network.forward(debugging_input_tensor);

        // Debugging output
        std::cout << "Policy Output: " << policy_output << std::endl;
        std::cout << "Target Output: " << target_output << std::endl;

        // Verify if weights match
        if (target_network.compare_parameters(policy_network, target_network)) {
            std::cout << "Target network successfully copied from policy network." << std::endl;
        } else {
            std::cerr << "Error: Target network parameters do not match the policy network." << std::endl;
        }

        // Verify if outputs match
        if (torch::allclose(policy_output, target_output, 1e-5, 1e-8)) {
            std::cout << "Outputs match between the policy network and the target network." << std::endl;
        } else {
            std::cerr << "Outputs do NOT match between the policy network and the target network." << std::endl;
        }
    }
}

//used to be based on an interval, now it just does what is asked.
void TrainingLoop::save_model(PolicyNetwork& policy_network,
                              const std::string& save_dir,
                              const std::string& run_id,
                              int global_frame_counter) {

    std::string save_path = save_dir + "/policy_network_" + run_id + "_" + std::to_string(global_frame_counter) + ".pt";

    // Save the model
    policy_network.save_model(save_path);

    // Log the save action
    std::cout << "Model saved at frame: " << global_frame_counter << " to " << save_path << std::endl;
}


void TrainingLoop::train_if_ready(int global_frame_counter, int training_interval, ReplayBuffer& replay_buffer,
                                  int batch_size, TargetNetwork& target_network) {
    // Check if it's time to train and if the replay buffer is ready
    if (global_frame_counter % training_interval == 0 && replay_buffer.is_ready(batch_size)) {
        train_step(target_network, replay_buffer, batch_size); // Perform the training step
    }
}