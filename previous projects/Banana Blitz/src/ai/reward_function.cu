#include "hip/hip_runtime.h"
#include "reward_function.hpp"
#include "player.hpp"
#include "environmental_object.hpp"
#include "collectable.hpp"
#include <iostream>
#include <cmath> 

RewardFunction::RewardFunction() {
    // Initialize previous positions to an invalid value, like -1, to detect the first frame
    prev_x_position = -1.0f;
    prev_y_position = -1.0f;

}

// Calculate rewards based on the current state of the game environment. Called every frame - is thus reset to 0 at the top of the function.
//Debugging outputs are included - comment in or out as needed.
float RewardFunction::calculate_reward(const player_class &player, int bananas_collected, float delta_time, bool level_completed, bool time_exceeded, float acceleration_factor) {
    float reward = 0.0f;

    float adjusted_delta_time = delta_time * acceleration_factor;


    // Penalty for Time Elapsing - To help incentize speed. Linear.
    float time_penalty = adjusted_delta_time * 1.0f;  // 1.0 points per second loss
    reward -= time_penalty;
    //std::cout << "Time penalty applied: -" << time_penalty << " points" << std::endl;

    // Reward for Banana Collection...
    if (bananas_collected > 0) {
        reward += bananas_collected * 50.0f;  // +50 points for each collected banana
        //std::cout << "Reward for collecting banana: +" << reward << " points" << std::endl;
    }

    //Penalty for object collision (dont know if this works yet.)
    if (player.get_is_colliding_with_platform()) {
        reward -= 0.01f;
        //std::cout << "Penalty for collision applied." << endl;
    }

    // Penalty for Time Exceeded - Should not be too large, because every episode is going to fail like this until the agent learns to perform so it is essentially meaningless
    if (time_exceeded) {
        reward -= 10.0f;  // -10 points for exceeding the maximum time without collecting a banana
        //std::cout << "Life clock expired penalty applied: -10 points (from reward_function.cu)" << std::endl;
    }

    // Reward for Level Completion
    if (level_completed) {
        reward += 1000.0f;  // +1000 points for completing the level
        //std::cout << "Reward for completing the level: +1000 points" << std::endl;
    }

    //Edge penalization logic
    const float edge_threshold = 51.0f; //pixels from the edge, change to 51 if no detection
    const float edge_penalty = 0.05f; //additional 5x the standard collision penalty

    float player_x = player.get_x_position();
    float player_y = player.get_y_position();
    const float player_size = 50.0f;

    if (player_x < (edge_threshold - player_size) || //left edge
        player_x > (1000.0f - edge_threshold) || //right edge
        player_y < (edge_threshold - player_size) || //top edge
        player_y > (800.0f - edge_threshold)) {
            reward -= edge_penalty;
            //std::cout << "Edge proximity detected and penalty applied: " << edge_penalty << " points." << endl;
        }

    //Movement Incentive logic

    if (prev_x_position >= 0.0f && prev_y_position >= 0.0f) {//this is always going to be true
        float movement_x = std::abs(player_x - prev_x_position);
        float movement_y = std::abs(player_y - prev_y_position);

        if (movement_x > 0.0f || movement_y > 0.0f) { //if any movement was done at all,
            const float movement_reward = 0.01f;
            reward += movement_reward;
            //std::cout << "Movement reward applied: + " << movement_reward << " points." << std::endl;
        
        }
    } 
    prev_x_position = player_x;
    prev_y_position = player_y;


    //next reward logic...

    return reward;
}
