#include "hip/hip_runtime.h"
#include "input_handler.hpp"
#include "object_type.hpp"
#include "raycasting.hpp"
#include "proximity_radius.hpp"
#include "player.hpp" // Include player header
#include <torch/torch.h>
#include <algorithm> // for padding or additional utility
#include <cmath> // for distance calculations
#include <cassert>
#include <cstdlib> //for rand() and RAND_MAX
#include <ctime>   // For seeding random numbers
#include <iostream>

InputHandler::InputHandler() {
    // Initialize any required data if necessary
}

//Note: this input handler handles both "inputs" in the sense that the player is inputting a move and getting an action performed back and also "inputs" in the sense of what mr. monkey's organs tell him about the world. I just got lazy and didn't want to make another file because i wouldnt know what to call it.


void InputHandler::handle_actions(player_class &player, float delta_time, const std::vector<environmental_object>& platforms, const std::vector<std::unique_ptr<collectable>>& bananas, float acceleration_factor) {
    bool player_up = false, player_down = false, player_left = false, player_right = false;

    if (is_ai_controlled) {
        // Get the action index from the policy network (assume this is passed or stored globally)
        int action_index = get_current_action_index(); // This is set by the DQN logic in the main loop - from experience_manager -> 

        // Map the action index to specific movement
        switch (action_index) {
            case 0: // Idle
                player_up = false;
                player_down = false;
                player_left = false;
                player_right = false;
                break;
            case 1: // Move up
                player_up = true;
                break;
            case 2: // Move down
                player_down = true;
                break;
            case 3: // Move left
                player_left = true;
                break;
            case 4: // Move right
                player_right = true;
                break;
            default:
                std::cerr << "Invalid action index: " << action_index << std::endl;
                break;
        }
    } else {
        // Human control mode: handle keyboard input
        player_up = sf::Keyboard::isKeyPressed(sf::Keyboard::Up);
        player_down = sf::Keyboard::isKeyPressed(sf::Keyboard::Down);
        player_left = sf::Keyboard::isKeyPressed(sf::Keyboard::Left);
        player_right = sf::Keyboard::isKeyPressed(sf::Keyboard::Right);
    }

    // Update the player's position based on the inputs
    player.update(player_up, player_down, player_left, player_right, delta_time, platforms, bananas, acceleration_factor);
}






// Setter for action index
void InputHandler::set_current_action_index(int index, float current_epsilon) {
    int post_epsilon_action = apply_epsilon(index, current_epsilon); // Apply epsilon-greedy logic
    action_index = post_epsilon_action; // Store the final action index after exploration consideration
    update_epsilon(); // Update epsilon after choosing the action
}

// Apply epsilon to the index
int InputHandler::apply_epsilon(int index, float current_epsilon) {
    // Generate a random float between 0 and 1 to decide if exploration occurs
    if (static_cast<float>(rand()) / static_cast<float>(RAND_MAX) < current_epsilon) {
        // Exploration: return a random action (0 to 4 if 5 total actions)
        return std::rand() % 5;
    } else {
        // Exploitation: return the action with the highest Q-value (the provided index)
        return index;
    }
}


void InputHandler::update_epsilon() {
// Decay epsilon but ensure it doesn’t fall below a minimum value, like 0.01
    current_epsilon = std::max(0.0f, current_epsilon - fixed_epsilon_decay);
    //std::cout << "Current epsilon is: " << current_epsilon << std::endl;
}

float InputHandler::get_current_epsilon() const {
    return current_epsilon;
}

// Getter for action index
int InputHandler::get_current_action_index() const {
    return action_index;
}


void InputHandler::set_ai_controlled(bool is_ai) {
    is_ai_controlled = is_ai;
}

// Function to gather raycasting data
std::vector<float> InputHandler::get_ray_inputs(const Raycasting &raycasting) {
    std::vector<float> ray_data;

    // Retrieve ray collisions and rays
    const auto& ray_collisions = raycasting.get_ray_collisions();
    const auto& rays = raycasting.get_rays();

    for (size_t i = 0; i < rays.size(); ++i) { // for all 20 rays
        const auto& ray = rays[i];
        float distance = std::hypot(ray[1].position.x - ray[0].position.x, ray[1].position.y - ray[0].position.y);

        // Get the collision type from the ray_collisions vector
        ObjectType collision_type = ray_collisions[i];

        // Push ray data: distance, collision type, endpoint x, endpoint y. This represents the 4 in the 4*20 inputs.
        ray_data.push_back(distance);
        ray_data.push_back(static_cast<float>(collision_type)); // Encode object type numerically. Values are in object_type.hpp, 0 for none, 1 for banana, 2 for platform
        ray_data.push_back(ray[1].position.x); // endpoint x
        ray_data.push_back(ray[1].position.y); // endpoint y
    }
    return ray_data;
}

// Function to gather proximity radius data
std::vector<float> InputHandler::get_proximity_inputs(const ProximityRadius &proximity_radius) {
    std::vector<float> proximity_data;

    for (const auto &object : proximity_radius.get_nearby_objects()) {
        proximity_data.push_back(object.distance);
        proximity_data.push_back(static_cast<float>(object.interaction_type)); // Use ObjectType directly as a float value
        proximity_data.push_back(object.nearest_point.x);
        proximity_data.push_back(object.nearest_point.y);
    }

    pad_proximity_data(proximity_data); // Pad if there are fewer objects than expected

    return proximity_data;
}

// Function to pad proximity data if fewer objects are detected
void InputHandler::pad_proximity_data(std::vector<float> &proximity_data) {
    constexpr int max_slots = 10; // Maximum number of proximity slots
    constexpr int features_per_slot = 4;

    while (proximity_data.size() < max_slots * features_per_slot) {
        // Pad with -1.0f to maintain fixed size, representing no nearby object
        proximity_data.push_back(-1.0f); // distance
        proximity_data.push_back(-1.0f); // interaction type
        proximity_data.push_back(-1.0f); // nearest_point.x
        proximity_data.push_back(-1.0f); // nearest_point.y
    }
}

// Function to gather player-specific inputs we don't need the whole object, just the position
std::vector<float> InputHandler::get_player_inputs(const player_class &player) {
    // Create a vector to store player-specific data
    std::vector<float> player_data;

    // Extract the x and y positions from the player using the getter methods
    player_data.push_back(player.get_x_position());
    player_data.push_back(player.get_y_position());

    return player_data;
}

// Function to normalize proximity radius inputs
void InputHandler::normalize_proximity_radius_inputs(std::vector<float> &proximity_data) {
    constexpr int features_per_slot = 4;
    for (size_t i = 0; i < proximity_data.size(); i += features_per_slot) {
        if (proximity_data[i] != -1.0f) { // Normalize only if it's not a padded value
            // Normalize distance assuming max possible distance is the radius itself
            proximity_data[i] /= 150.0f; // Assuming 150 is the maximum radius

            // Normalize interaction type 0 = None, 1 = Banana, 2 = Platform
            proximity_data[i + 1] /= 2.0f;

            // Normalize nearest_point.x and nearest_point.y assuming window size is 1000x800
            proximity_data[i + 2] /= 1000.0f;
            proximity_data[i + 3] /= 800.0f;
        }
    }
}

void InputHandler::normalize_raycasting_inputs(std::vector<float> &ray_data) {
    constexpr int features_per_ray = 4;
    
    // Ensure that the ray_data is a multiple of features_per_ray
    assert(ray_data.size() % features_per_ray == 0 && "Ray data size is not a multiple of features_per_ray");

    for (size_t i = 0; i < ray_data.size(); i += features_per_ray) {
        if (ray_data[i] != -1.0f) { // Normalize only if it's not a padded value
            // Normalize distance (assuming max ray length is 1500
            ray_data[i] /= 1500.0f;

            // Normalize collision type (0 = None, 1 = Banana, 2 = Platform
            ray_data[i + 1] /= 2.0f;

            // Normalize endpoint x and y (assuming window size is 1000x800
            ray_data[i + 2] /= 1000.0f;
            ray_data[i + 3] /= 800.0f;
        }
    }
}

// Function to normalize player-specific inputs
void InputHandler::normalize_player_inputs(std::vector<float> &player_data) {
    // Normalize player x and y positions (assuming window size is 1000x800
    player_data[0] /= 1000.0f;
    player_data[1] /= 800.0f;
}

// Convert combined input vector to a Torch tensor
torch::Tensor InputHandler::convert_to_tensor(const std::vector<float>& combined_inputs) {
    // Convert std::vector<float> to torch::Tensor and move to GPU
    torch::Tensor tensor = torch::from_blob(
        const_cast<float*>(combined_inputs.data()), 
        {1, static_cast<long>(combined_inputs.size())}, 
        torch::kFloat
    ).clone().to(torch::kCUDA); // clone to copy data since from_blob just references the original, and move to GPU

    return tensor;
}

// Function to combine all inputs and directly convert to a Torch tensor
torch::Tensor InputHandler::get_combined_inputs(const Raycasting &raycasting, const ProximityRadius &proximity_radius, const player_class &player) {
    // Gather inputs from different parts of the game environment
    std::vector<float> ray_data = get_ray_inputs(raycasting);
    std::vector<float> proximity_data = get_proximity_inputs(proximity_radius);
    std::vector<float> player_data = get_player_inputs(player);

    // Normalize each set of inputs
    normalize_raycasting_inputs(ray_data);
    normalize_proximity_radius_inputs(proximity_data);
    normalize_player_inputs(player_data);

    // Combine all inputs into one vector
    std::vector<float> combined_inputs;
    combined_inputs.reserve(ray_data.size() + proximity_data.size() + player_data.size());  // Reserve space to improve efficiency

    combined_inputs.insert(combined_inputs.end(), ray_data.begin(), ray_data.end());
    combined_inputs.insert(combined_inputs.end(), proximity_data.begin(), proximity_data.end());
    combined_inputs.insert(combined_inputs.end(), player_data.begin(), player_data.end());

    // Set negative values to zero only if they were introduced due to a calculation error (not for padding values
    for (auto &value : combined_inputs) {
        if (value < 0.0f && value != -1.0f) {
            value = 0.0f;
        }
    }

    // Convert combined input vector to a Torch tensor and move to GPU
    torch::Tensor input_tensor = torch::from_blob(
        combined_inputs.data(),
        {1, static_cast<long>(combined_inputs.size())},
        torch::kFloat
    ).clone().to(torch::kCUDA); // clone to copy data since from_blob just references the original, and move to GPU
    
    /*
    // Debugging output to check if the tensor is on GPU
    if (input_tensor.device().is_cuda()) {
        std::cout << "Tensor is on GPU." << std::endl;
    } else {
        std::cout << "Tensor is on CPU. CUDA move failed." << std::endl;
    }
    */
    return input_tensor;
}


torch::Tensor InputHandler::generate_debug_input() {
    // Generate a random tensor with the same shape as your state input
    torch::Tensor debug_tensor = torch::rand({1, 122}, torch::kFloat32); // Adjust size to match your state tensor
    if (torch::cuda::is_available()) {
        debug_tensor = debug_tensor.to(torch::kCUDA);
    }
    return debug_tensor;
}
